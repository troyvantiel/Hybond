#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <gpuheader.h>
#include <gpuvdwdirectp_kernel.cu>
////////////////////////////////////////////////////////////////////////////////

extern "C"
void gpuvdwdirectp_ 
(double*, int, int*, int, double*, double*, int, double*, int, int, double*, double*, int, int, int);
void gpuvdwdirectp__
(double*, int, int*, int, double*, double*, int, double*, int, int, double*, double*, int, int, int);

extern "C"
void gpuvdwdirectp_
(double* x, int num_atmall, int* atype, int nat, double* epsilon4, double* sigma2, int tblno,
 double* xmax, int periodicflag, int natchangeflag, double* force, double* pot, int idevice,
 int begin_numi, int num_atmi)
{
  CUDA_SAFE_CALL(hipSetDevice(idevice));
  int itmp, itmp1;
   int tex_l = 512;
   int num_atm;
   int num_site;
   num_site = 1;
   num_atm = num_atmall / num_site;
   int num_atm_i = num_atmi;
   int iatm_begin = begin_numi - 1;
   int iatm_end = iatm_begin + num_atm_i - 1;
   if (num_site != 1){
     if (iatm_begin != 0)
       iatm_begin = 1+(int)((float)(iatm_begin-1)/(float)num_site);
     iatm_end = (int)((float)(iatm_end)/(float)num_site);
     num_atm_i = iatm_end - iatm_begin + 1;
   }
   int n  = ((int)((float)(num_atm-1)/(float)BVDWDIRC)+1)*BVDWDIRC;
   int ni = ((int)((float)(num_atm_i-1)/(float)BVDWDIRC)+1)*BVDWDIRC;
   float xmax_float0 = (float)xmax[0];
   float xmax_float1 = (float)xmax[1];
   float xmax_float2 = (float)xmax[2];
   float rcp_xmax0 = (float)(1.e0/xmax[0]);
   float rcp_xmax1 = (float)(1.e0/xmax[1]);
   float rcp_xmax2 = (float)(1.e0/xmax[2]);
   int thd2 = (int)sqrt((float)(THD/4));
   if (thd2 < nat)
     printf("WARNING:gpuvdwforce256 TOO MUCH NUMBER OF ATM TYPE");
   unsigned int memsize_x_float = sizeof(vdwatm) * n;
   vdwatm* x_float = (vdwatm*) malloc(memsize_x_float);
   unsigned int memsize_xi_float = sizeof(vdwatm) * ni;
   vdwatm* xi_float = (vdwatm*) malloc(memsize_xi_float);
   unsigned int memsize_gr_float = sizeof(float4) * THD/4;
   float4* gr_float = (float4*) malloc(memsize_gr_float);
   unsigned int memsize_f_float = sizeof(float4) * ni;
   float4* f_float = (float4*) malloc(memsize_f_float);
   unsigned int memsize_tex = sizeof(float4) * tex_l;
   float4* tex_data = (float4*) malloc(memsize_tex);
   
   for (int i = 0; i < num_atm; i++){
     COPY_DOUBLE_TO_VDWATM(x_float,x,atype,i,i*num_site)
   }
   for (int i = num_atm  ; i < n        ; i++){
     SET_DUMMY_VDWATM(x_float[i],nat)
   }
   for (int i = 0        ; i < num_atm_i; i++){
     xi_float[i] = x_float[i+iatm_begin];
   }
   for (int i = num_atm_i; i < ni       ; i++){
     SET_DUMMY_VDWATM(xi_float[i],nat)
   }
   for (int i = 0        ; i < THD/4    ; i++){
     SET_FLOAT4_ZERO(gr_float[i])
   }
   for (int i = 0; i < nat; i++)
     for (int j = 0; j < nat; j++){
       itmp = i*(nat+1)+j;
       itmp1 = i*(nat)+j;
       COPY_DOUBLE_TO_VDWCOEF(gr_float,epsilon4,sigma2,itmp,itmp1)
     }
   
   CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_B), gr_float , memsize_gr_float) );
   vdwatm* x_dev;
   CUDA_SAFE_CALL(hipMalloc((void**)&x_dev, memsize_x_float));
   CUDA_SAFE_CALL(hipMemcpy(x_dev, x_float  , memsize_x_float, hipMemcpyHostToDevice) );
   vdwatm* xi_dev;
   CUDA_SAFE_CALL(hipMalloc((void**)&xi_dev, memsize_xi_float));
   CUDA_SAFE_CALL(hipMemcpy(xi_dev, xi_float  , memsize_xi_float, hipMemcpyHostToDevice) );
   float4* f_dev;
   CUDA_SAFE_CALL(hipMalloc((void**)&f_dev, memsize_f_float));
   dim3 threads(BVDWDIRC);
   dim3 grid(ni / BVDWDIRC);
   int nat2 = nat + 1;
   gpuvdwdirectp_kernel<<< grid, threads >>>
     (f_dev, x_dev, xi_dev, n, xmax_float0, xmax_float1, xmax_float2, nat2,rcp_xmax0,rcp_xmax1,rcp_xmax2);
   CUT_CHECK_ERROR("Kernel execution failed");
   CUDA_SAFE_CALL(hipMemcpy(f_float, f_dev, memsize_f_float,hipMemcpyDeviceToHost) );

#if 0
   double dx[6];
   for (int i = 0; i < num_atm; ++i){
     pot[i] = 0.e0;
     for (int k = 0; k < 3; ++k)
       force[i*3+k] = 0.e0;
     for (int j = 0; j < num_atm; ++j){
       for (int k = 0; k < 3; ++k){
	 dx[k] = x[i*3+k] - x[j*3+k];
	 //dx[k] -= rint(dx[k]/xmax[k])*xmax[k];
       }
       dx[3] = dx[0]*dx[0]+dx[1]*dx[1]+dx[2]*dx[2];
       dx[3] += 2.e0;
       //if (dx[3] != 0.e0){
	 dx[3] = 1.e0/dx[3];
	 dx[4] = dx[3]*dx[3]*dx[3];
	 pot[i] += 4.e0 * dx[4] * (dx[4] - 1.e0);
	 dx[5] = 24.e0 * dx[3] * dx[4] * (2.e0 * dx[4] - 1.e0);
	 for (int k = 0; k < 3; ++k)
	   force[i*3+k] += dx[5] * dx[k];
	 //}
     }
   }
   double sum[3]={0.0,0.0,0.0};
   for (int i = 0; i < num_atm; ++i)
     for (int k = 0; k < 3; ++k)
       sum[k] += force[i*3+k]*force[i*3+k];
#endif 
   
   for (int i = 0; i < num_atm_i; ++i){
     itmp = (i+iatm_begin)*num_site;
     COPY_SINGLE_TO_FORCEPOT(f_float,force,pot,itmp,i)
   }
#if 0
   double sum0[3]={0.0,0.0,0.0};
   for (int i = 0; i < num_atm; ++i)
     for (int k = 0; k < 3; ++k)
       sum0[k] += force[i*3+k]*force[i*3+k];
   //printf("host %f %f %f\n",sum[0],sum[1],sum[2]);
   //printf("gpu  %f %f %f\n",sum0[0],sum0[1],sum0[2]);
#endif
   
   free(x_float);
   free(xi_float);
   free(f_float);
   free(gr_float);
   free(tex_data);
   CUDA_SAFE_CALL(hipFree(x_dev));
   CUDA_SAFE_CALL(hipFree(xi_dev));
   CUDA_SAFE_CALL(hipFree(f_dev));
}

extern "C"
void
gpuvdwdirectp__
(double* x, int* n, int* atype, int* nat, double* epsilon4, double* sigma2, int* tblno,
 double* xmax, int* periodicflag, int* natchangeflag, double* force, double* pot, int* idevice,
 int* begin_numi, int* num_atmi)
{
  gpuvdwdirectp_
    (x,*n,atype,*nat,epsilon4,sigma2,*tblno,xmax,*periodicflag,*natchangeflag,force,pot,*idevice,
     *begin_numi,*num_atmi);
}
