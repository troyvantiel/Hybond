#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <gpuheader.h>
#include <hipfft/hipfft.h>
extern "C"
void gpufft3d_  (float*, int, int, int, int, int);
void gpufft3d__ (float*, int, int, int, int, int);
extern "C"
void
gpufft3d_ (float* x, int size_x, int size_y, int size_z, int fftflag, int idevice)
{
   /* CUDA_SAFE_CALL(hipSetDevice(idevice)); */
   unsigned int mem_size_A = sizeof(hipfftComplex) * size_x * size_y * size_z;
   hipfftHandle plan;
   hipfftComplex *idata, *odata;
   CUDA_SAFE_CALL(hipMalloc((void**) &idata, mem_size_A));
   CUDA_SAFE_CALL(hipMalloc((void**) &odata, mem_size_A));
   CUDA_SAFE_CALL(hipMemcpy(idata, x, mem_size_A, hipMemcpyHostToDevice));
   hipfftPlan3d(&plan, size_x, size_y, size_z, HIPFFT_C2C);
   if (fftflag == 1){
     hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
   }
   if (fftflag == -1){
     hipfftExecC2C(plan, idata, odata, HIPFFT_BACKWARD);
   }
   CUDA_SAFE_CALL(hipMemcpy(x, odata, mem_size_A,hipMemcpyDeviceToHost));
   hipfftDestroy(plan);
   CUDA_SAFE_CALL(hipFree(idata));
   CUDA_SAFE_CALL(hipFree(odata));
}
extern "C"
void
gpufft3d__ (float* x, int* size_x, int* size_y, int* size_z, int* flagfft, int* idevice)
{
  gpufft3d_ (x, *size_x, *size_y, *size_z, *flagfft, *idevice);
}
