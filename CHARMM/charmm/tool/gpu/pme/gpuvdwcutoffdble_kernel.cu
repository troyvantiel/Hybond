#include "hip/hip_runtime.h"
#include <stdio.h>
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#define DS(i, j) Ds[i][j]
#define ES(i, j) Es[i][j]
#define FS(i, j) Fs[i][j]
#define FORCELOOP(iatm)				\
  dx[0] = xi.x - BS(0,iatm).x;			\
  dx[1] = xi.y - BS(0,iatm).y;			\
  dx[2] = xi.z - BS(0,iatm).z;			\
  dx[0] -= rintf(dx[0] * al0) * xmax0;		\
  dx[1] -= rintf(dx[1] * al1) * xmax1;		\
  dx[2] -= rintf(dx[2] * al2) * xmax2;		\
  dx[3] = dx[0]*dx[0];				\
  dx[3] += dx[1]*dx[1];				\
  dx[3] += dx[2]*dx[2];				\
  if (dx[3] < cut2){				\
    if (dx[3] != 0.f){				\
      dx[3] = 1.f / dx[3];			\
    }else{					\
      dx[3] = 0.f;				\
    }						\
    itmp = xi.a + BS(0,iatm).a;					\
    coef = d_B[itmp];						\
    dn6 = dx[3]*dx[3]*dx[3];					\
    tmp0 = dn6 * (coef.x * dn6 - coef.y);			\
    ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub2.z,Csub2.w)	\
      dn6 = dx[3] * dn6 * (coef.z * dn6 - coef.w);		\
    tmp0 = dn6 * dx[0];						\
    ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub1.x,Csub1.y)	\
      tmp0 = dn6 * dx[1];					\
    ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub1.z,Csub1.w)	\
      tmp0 = dn6 * dx[2];					\
    ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub2.x,Csub2.y)	\
	}
texture<float4,1,hipReadModeElementType> tex;
__constant__ float4 d_B[THD/4];
__shared__ vdwatm Bs[1][BVDWDIRC];

extern "C"
__global__ void gpuvdwcutoffdble_kernel
(float4* C, vdwatm* A, vdwatm* Ai, int num, float xmax0, float xmax1, float xmax2,
 int nat, float al0, float al1, float al2, float cut2)
{
  int ty = threadIdx.x;
  int itmp;
  float4 Csub1, Csub2, coef;
  vdwatm xi;
  float dx[4], dn6, tmp0, tmp1, tmp2, tmp3;

  SET_FLOAT4_ZERO(Csub1)
  SET_FLOAT4_ZERO(Csub2)
  xi = Ai[BVDWDIRC*blockIdx.x+ty];
  xi.a *= nat;
  for (int j = 0; j < num/BVDWDIRC; j++){
    BS(0,ty) = A[j*BVDWDIRC+ty];
    __syncthreads();
    for (int i = 0; i < BVDWDIRC; i++){
      FORCELOOP(i)
    }
    /*    for (int i = 0; i < BVDWDIRC/8; i++){
      FORCELOOP(i*8+0)
      FORCELOOP(i*8+1)
      FORCELOOP(i*8+2)
      FORCELOOP(i*8+3)
      FORCELOOP(i*8+4)
      FORCELOOP(i*8+5)
      FORCELOOP(i*8+6)
      FORCELOOP(i*8+7)
      }*/
    __syncthreads();
  }
  C[BVDWDIRC*blockIdx.x*2+ty*2+0] = Csub1;
  C[BVDWDIRC*blockIdx.x*2+ty*2+1] = Csub2;
}

