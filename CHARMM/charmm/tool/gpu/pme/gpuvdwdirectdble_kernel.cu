#include "hip/hip_runtime.h"
#include <stdio.h>
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#define DS(i, j) Ds[i][j]
#define ES(i, j) Es[i][j]
#define FS(i, j) Fs[i][j]
texture<float4,1,hipReadModeElementType> tex;
__constant__ float4 d_B[THD/4];
__shared__ vdwatm Bs[1][BVDWDIRC];

extern "C"
__global__ void gpuvdwdirectdble_kernel
(float4* C, vdwatm* A, vdwatm* Ai, int num, float xmax0, float xmax1, float xmax2, int nat,
 float al0, float al1, float al2 )
{
  int ty = threadIdx.x;
  int itmp;
  float4 Csub1, Csub2, coef;
  vdwatm xi;
  float dx[4], dn6, tmp0, tmp1, tmp2, tmp3;

  SET_FLOAT4_ZERO(Csub1)
  SET_FLOAT4_ZERO(Csub2)
  xi = Ai[BVDWDIRC*blockIdx.x+ty];
  xi.a *= nat;
  for (int j = 0; j < num/BVDWDIRC; j++){
    BS(0,ty) = A[j*BVDWDIRC+ty];
    __syncthreads();
    for (int i = 0; i < BVDWDIRC; i++){
      dx[0] = xi.x - BS(0,i).x;
      dx[1] = xi.y - BS(0,i).y;
      dx[2] = xi.z - BS(0,i).z;
      dx[0] -= rintf(dx[0] * al0) * xmax0;
      dx[1] -= rintf(dx[1] * al1) * xmax1;
      dx[2] -= rintf(dx[2] * al2) * xmax2;
      dx[3] = dx[0]*dx[0];
      dx[3] += dx[1]*dx[1];
      dx[3] += dx[2]*dx[2];
      if (dx[3] != 0.f){
      	dx[3] = 1.f / dx[3];
      }else{
      	dx[3] = 0.f;
      }
      itmp = xi.a + BS(0,i).a;
      coef = d_B[itmp];
      dn6 = dx[3]*dx[3]*dx[3];
      tmp0 = dn6 * (coef.x * dn6 - coef.y);
      ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub2.z,Csub2.w)
      //Csub.w += dn6 * (coef.x * dn6 - coef.y);
      dn6 = dx[3] * dn6 * (coef.z * dn6 - coef.w);
      tmp0 = dn6 * dx[0];
      ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub1.x,Csub1.y)
      tmp0 = dn6 * dx[1];
      ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub1.z,Csub1.w)
      tmp0 = dn6 * dx[2];
      ADD_DOUBLEFLOAT(tmp0,tmp1,tmp2,tmp3,Csub2.x,Csub2.y)
      //Csub.x += dn6 * dx[0];
      //Csub.y += dn6 * dx[1];
      //Csub.z += dn6 * dx[2];
    }
    __syncthreads();
  }
  C[BVDWDIRC*blockIdx.x*2+ty*2+0] = Csub1;
  C[BVDWDIRC*blockIdx.x*2+ty*2+1] = Csub2;
}

