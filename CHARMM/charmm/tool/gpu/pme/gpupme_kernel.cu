#include "hip/hip_runtime.h"
#if 0 // not work, too much shared memory
#define QS_SIZEX 16
#define QS_SIZEY 16
#define QS_SIZEZ 16
#elif 0 // supports 9 grids per block and order=6, work
#define QS_SIZEX 15
#define QS_SIZEY 15
#define QS_SIZEZ 15
#else // supports 8 grids per block and order=6, work
#define QS_SIZEX 14
#define QS_SIZEY 14
#define QS_SIZEZ 14
#endif
#define QS_SIZE  QS_SIZEX*QS_SIZEY*QS_SIZEZ

#ifdef FIXEDPOINT
#define INT_OR_FLOAT   int
#else
#define INT_OR_FLOAT   float
#endif

texture<float2,1,hipReadModeElementType> tex;
extern "C"
__global__ void gpupme1_kernel
(float* q_dev, float* bx_dev, float* by_dev, float* bz_dev, int sizex,
 int sizey, int sizez, float* recipf_dev, float factor, float density)
{
  float m2, tmp, eterm;
  float mx, my, mz, q[2];
  //int nx = blockIdx.x * BSIZEPME1 + threadIdx.x;
  //int ny = blockIdx.y * BSIZEPME1 + threadIdx.y;
  //int nz = blockIdx.z * BSIZEPME1 + threadIdx.z;
  //int num = nx + ny * sizex + nz * sizex * sizey;
#if 1 // modified by Ryuji, Nov. 2011
  int num = (blockIdx.y+blockIdx.x*gridDim.y) * BSIZEPME1 + threadIdx.x;
  if (num >= sizex*sizey*sizez) return;
#else
  int num = blockIdx.x * BSIZEPME1 + threadIdx.x;
#endif
  int nz = (int)((float)(num)/(float)(sizex*sizey));
  int ny = (int)((float)(num-nz*sizex*sizey)/(float)(sizex));
  int nx = num-nz*sizex*sizey-ny*sizex;
  if (2*nx > sizex){ mx = (float)(nx-sizex);
  }else{             mx = (float)(nx);      }
  if (2*ny > sizey){ my = (float)(ny-sizey);
  }else{             my = (float)(ny);      }
  if (2*nz > sizez){ mz = (float)(nz-sizez);
  }else{             mz = (float)(nz);      }
#if 0
  if(blockIdx.x==0 && threadIdx.x==0) printf("****\n");
  if(nx<0 || num>=sizex*sizey*sizez || mx<0 || my<0 || mz<0){
    printf("bid=%d tid=%d nx=%d num=%d mx,y,z=%d %d %d\n",
	   blockIdx.x,threadIdx.x,nx,num,mx,my,mz);
  }
#endif
  //  if(num<0 || num>=sizex*sizey*sizez){
  //      printf("bid=%d tid=%d num=%d\n",blockIdx.x,threadIdx.x,num);
  //  }
  q[0] = q_dev[num*2+0];
  q[1] = q_dev[num*2+1];
  tmp = mx*recipf_dev[0]+my*recipf_dev[3]+mz*recipf_dev[6];
  m2 = tmp*tmp;
  tmp = mx*recipf_dev[1]+my*recipf_dev[4]+mz*recipf_dev[7];
  m2 += tmp*tmp;
  tmp = mx*recipf_dev[2]+my*recipf_dev[5]+mz*recipf_dev[8];
  m2 += tmp*tmp;
  if (num > 0 && num < sizex*sizey*sizez){
    tmp = 1.f/m2;
  }else{
    tmp = 0.f;    
  }
  tmp *= density;
  //  if(nx<0 || nx>=sizex || ny<0 || ny>=sizey || nz<0 || nz>=sizez){
  //      printf("bid=%d tid=%d num=%d nx,y,z=%d %d %d\n",blockIdx.x,threadIdx.x,num,nx,ny,nz);
  //  }
  tmp *= bx_dev[nx];
  tmp *= by_dev[ny];
  tmp *= bz_dev[nz];
  eterm = expf(-factor*m2)*tmp; 
  q_dev[num*2+0] = q[0] * eterm;
  q_dev[num*2+1] = q[1] * eterm;
}

// gpupme1_kernel_stress is written by Ryuji on Oct. 29, 2010.
extern "C"
__global__ void gpupme1_kernel_stress
(float* q_dev, float* bx_dev, float* by_dev, float* bz_dev, int sizex,
 int sizey, int sizez, float* recipf_dev, float factor, float density,
 float* stress_dev)
{
  __shared__ volatile float stress_shd[6*BSIZEPME1];
  float mx, my, mz, q[2];
  //int nx = blockIdx.x * BSIZEPME1 + threadIdx.x;
  //int ny = blockIdx.y * BSIZEPME1 + threadIdx.y;
  //int nz = blockIdx.z * BSIZEPME1 + threadIdx.z;
  //int num = nx + ny * sizex + nz * sizex * sizey;
#if 1 // modified by Ryuji, Nov. 2011
  int num = (blockIdx.y + blockIdx.x * gridDim.y) * BSIZEPME1 + threadIdx.x;
  if (num >= sizex*sizey*sizez) return;
#else
  int num = blockIdx.x * BSIZEPME1 + threadIdx.x;
#endif
  int nz = (int)((float)(num)/(float)(sizex*sizey));
  int ny = (int)((float)(num-nz*sizex*sizey)/(float)(sizex));
  int nx = num-nz*sizex*sizey-ny*sizex;
  if (2*nx > sizex){ mx = (float)(nx-sizex);
  }else{             mx = (float)(nx);      }
  if (2*ny > sizey){ my = (float)(ny-sizey);
  }else{             my = (float)(ny);      }
  if (2*nz > sizez){ mz = (float)(nz-sizez);
  }else{             mz = (float)(nz);      }
  q[0] = q_dev[num*2+0];
  q[1] = q_dev[num*2+1];
  float m0 = mx*recipf_dev[0]+my*recipf_dev[3]+mz*recipf_dev[6];
  float m1 = mx*recipf_dev[1]+my*recipf_dev[4]+mz*recipf_dev[7];
  float m2 = mx*recipf_dev[2]+my*recipf_dev[5]+mz*recipf_dev[8];
  float msq = m0*m0 + m1*m1 + m2*m2;
  float rmsq = 0.f;	
  if (num > 0 && num < sizex*sizey*sizez) rmsq = 1.f/msq;
  float ptmp = 2.f * (factor + rmsq);
  float eterm = expf(-factor*msq) * rmsq * density * bx_dev[nx] * by_dev[ny] * bz_dev[nz];
  float sn = eterm * (q[0]*q[0] + q[1]*q[1]);
  stress_shd[threadIdx.x*6+0] = sn * (- ptmp * m0 * m0 + 1.f);
  stress_shd[threadIdx.x*6+1] = sn * (- ptmp * m0 * m1);
  stress_shd[threadIdx.x*6+2] = sn * (- ptmp * m0 * m2);
  stress_shd[threadIdx.x*6+3] = sn * (- ptmp * m1 * m1 + 1.f);
  stress_shd[threadIdx.x*6+4] = sn * (- ptmp * m1 * m2);
  stress_shd[threadIdx.x*6+5] = sn * (- ptmp * m2 * m2 + 1.f);
   __syncthreads();
  q_dev[num*2+0] = q[0] * eterm;
  q_dev[num*2+1] = q[1] * eterm;
  for (unsigned int s = BSIZEPME1/2; s > 32; s >>= 1){
    if (threadIdx.x < s){
      stress_shd[threadIdx.x*6+0] += stress_shd[(threadIdx.x+s)*6+0];
      stress_shd[threadIdx.x*6+1] += stress_shd[(threadIdx.x+s)*6+1];
      stress_shd[threadIdx.x*6+2] += stress_shd[(threadIdx.x+s)*6+2];
      stress_shd[threadIdx.x*6+3] += stress_shd[(threadIdx.x+s)*6+3];
      stress_shd[threadIdx.x*6+4] += stress_shd[(threadIdx.x+s)*6+4];
      stress_shd[threadIdx.x*6+5] += stress_shd[(threadIdx.x+s)*6+5];
    }
    __syncthreads();
  }
  if (threadIdx.x < 32)
    for (unsigned int s = 32; s > 0; s >>= 1){
      stress_shd[threadIdx.x*6+0] += stress_shd[(threadIdx.x+s)*6+0];
      stress_shd[threadIdx.x*6+1] += stress_shd[(threadIdx.x+s)*6+1];
      stress_shd[threadIdx.x*6+2] += stress_shd[(threadIdx.x+s)*6+2];
      stress_shd[threadIdx.x*6+3] += stress_shd[(threadIdx.x+s)*6+3];
      stress_shd[threadIdx.x*6+4] += stress_shd[(threadIdx.x+s)*6+4];
      stress_shd[threadIdx.x*6+5] += stress_shd[(threadIdx.x+s)*6+5];
    }
#if 1 // modified by Ryuji, Nov. 2011
  if (threadIdx.x < 6)
    stress_dev[(blockIdx.y + blockIdx.x * gridDim.y)*6+threadIdx.x] = stress_shd[threadIdx.x];
#else
  if (threadIdx.x < 6)
    stress_dev[blockIdx.x*6+threadIdx.x] = stress_shd[threadIdx.x];
#endif
}

extern "C"
__global__ void gpupme1gradsum_kernel
(float4* x, float4* f, float* q, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder,
 int (*cpb)[4], int gridperblockx, int gridperblocky, int gridperblockz)
{
  float ui[3], qtmp, utmp1, utmp0;
  float4 ftmp, xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0, mtmp2;
  int cpt=cpb[blockIdx.x][3] & 0xf;
  int cpbone[3]={(cpb[blockIdx.x][0] & 0xff)*gridperblockx,
		 (cpb[blockIdx.x][1] & 0xff)*gridperblocky,
		 (cpb[blockIdx.x][2] & 0xff)*gridperblockz};
  int i,cptoffset=cpt*sizex*sizey*sizez*2;
  int miorg[3],mioffset[3];
#ifdef MULTIPLE_IBLOCK
  int fb=cpb[blockIdx.x][0]>>8,nb=(cpb[blockIdx.x][1]>>8)-fb;
  int num = fb * BSIZEPME2 + threadIdx.x;
#else
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;
#endif
#define USE_SHARED_FOR_GRADSUM

  for(int k=0;k<3;k++) miorg[k]=cpbone[k];

#ifdef USE_SHARED_FOR_GRADSUM
  __shared__ float qs[QS_SIZE];
#if 0 // try to optimize, not work
  unsigned int ti;
  ti=threadIdx.x & 0xf;
  for(i=(threadIdx.x>>4)<<4;i<QS_SIZE;i+=BSIZEPME2){
    if(i<QS_SIZE){
      mtmp1=(i/QS_SIZEX) % QS_SIZEY;
      mtmp2=(i/QS_SIZEX/QS_SIZEY) % QS_SIZEZ;
      if(ti<QS_SIZEX){
	mtmp0=ti;
	mtmp0+=miorg[0];if(mtmp0>=sizex) mtmp0-=sizex;
	mtmp1+=miorg[1];if(mtmp1>=sizey) mtmp1-=sizey;
	mtmp2+=miorg[2];if(mtmp2>=sizez) mtmp2-=sizez;
	qs[i+ti]=q[2*(mtmp0+sizex*(mtmp1+sizey*mtmp2))];
      }
      __syncthreads();
    }
  }
  __syncthreads();
#else // work
  for(i=threadIdx.x;i<QS_SIZE;i+=BSIZEPME2){
    mtmp0=i % QS_SIZEX;
    mtmp1=(i/QS_SIZEX) % QS_SIZEY;
    mtmp2=(i/QS_SIZEX/QS_SIZEY) % QS_SIZEZ;
    mtmp0+=miorg[0];if(mtmp0>=sizex) mtmp0-=sizex;
    mtmp1+=miorg[1];if(mtmp1>=sizey) mtmp1-=sizey;
    mtmp2+=miorg[2];if(mtmp2>=sizez) mtmp2-=sizez;
    qs[i]=q[2*(mtmp0+sizex*(mtmp1+sizey*mtmp2))];
  }
  __syncthreads();
#endif
#else
  if(blockIdx.x==0 && threadIdx.x==0) printf("** USE_SHARED_FOR_GRADSUM is not defined **\n");
#endif

#ifdef MULTIPLE_IBLOCK
  for(int b=0;b<nb;b++,num+=BSIZEPME2){
#endif
  SET_FLOAT4_ZERO(ftmp);
  xi = x[num];
  if(1 && xi.w!=0.0f){
    //  if(1){
#if defined(CELLINDEX) && 1
    ui[0] = __fmul_rn(xi.x,recipf_dev[0])+__fmul_rn(xi.y,recipf_dev[1])+__fmul_rn(xi.z,recipf_dev[2]);
    ui[1] = __fmul_rn(xi.x,recipf_dev[3])+__fmul_rn(xi.y,recipf_dev[4])+__fmul_rn(xi.z,recipf_dev[5]);
    ui[2] = __fmul_rn(xi.x,recipf_dev[6])+__fmul_rn(xi.y,recipf_dev[7])+__fmul_rn(xi.z,recipf_dev[8]);
#else
    ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
    ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
    ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#endif
#if 1
    for(int j=0;j<3;j++){
      while(ui[j]<0.0f)  ui[j]+=1.0f;
      while(ui[j]>=1.0f) ui[j]-=1.0f;
    }
#endif                             
    ui[0] *= (float)sizex;
    ui[1] *= (float)sizey;
    ui[2] *= (float)sizez;
    mi[0] = (int)ui[0];
    mi[1] = (int)ui[1];
    mi[2] = (int)ui[2];
    ui[0] = - ui[0] + (float)mi[0];
    ui[1] = - ui[1] + (float)mi[1];
    ui[2] = - ui[2] + (float)mi[2];
    mi[0] += sizex - order;
    mi[1] += sizey - order;
    mi[2] += sizez - order;
    //    if(xi.w==0.0f) for(int k=0;k<3;k++) mi[k]=miorg[k]-1;
#ifdef USE_SHARED_FOR_GRADSUM // minimum image conversion
    while(mi[0]-miorg[0]>=sizex/2) mi[0]-=sizex;
    while(mi[1]-miorg[1]>=sizey/2) mi[1]-=sizey;
    while(mi[2]-miorg[2]>=sizez/2) mi[2]-=sizez;
    while(mi[0]-miorg[0]<-sizex/2) mi[0]+=sizex;
    while(mi[1]-miorg[1]<-sizey/2) mi[1]+=sizey;
    while(mi[2]-miorg[2]<-sizez/2) mi[2]+=sizez;
#else // added to ensure overflow of grids
    if(mi[0]+1>=sizex) mi[0]-=sizex;
    if(mi[1]+1>=sizey) mi[1]-=sizey;
    if(mi[2]+1>=sizez) mi[2]-=sizez;
    if(mi[0]+1<0)        mi[0]+=sizex;
    if(mi[1]+1<0)        mi[1]+=sizey;
    if(mi[2]+1<0)        mi[2]+=sizez;
#endif
    float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
    for (int k = 0; k < order; k++){
      ui[2] += 1.f;
      //      ttmp2 = tex1D(tex,ui[2]*rorder);
      ui2tex += rorder;
      ttmp2 = tex1D(tex,ui2tex);
      mi[2] += 1;
#ifndef USE_SHARED_FOR_GRADSUM
      if (mi[2] >= sizez) mi[2] -= sizez;
#endif
      mioffset[2]=mi[2]-miorg[2];//if(mioffset[2]<0) mioffset[2]+=QS_SIZEZ;if(mioffset[2]>=QS_SIZEZ) mioffset[2]-=QS_SIZEZ;
      utmp1 = ui[1];
      mtmp1 = mi[1];
      float utmp1tex=utmp1*rorder*(NSEG-1)/(float)NSEG;
      for (int j = 0; j < order; j++){
	utmp1 += 1.f;
	//	ttmp1 = tex1D(tex,utmp1*rorder);
	utmp1tex += rorder;
	ttmp1 = tex1D(tex,utmp1tex);
	mtmp1 += 1;
#ifndef USE_SHARED_FOR_GRADSUM
	if (mtmp1 >= sizey) mtmp1 -= sizey;
#endif
	mioffset[1]=mtmp1-miorg[1];//if(mioffset[1]<0) mioffset[1]+=QS_SIZEY;if(mioffset[1]>=QS_SIZEY) mioffset[1]-=QS_SIZEY;
	utmp0 = ui[0];
	mtmp0 = mi[0];
	float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
	for (int i = 0; i < order; i++){
	  utmp0 += 1.f;
	  //	  ttmp0 = tex1D(tex,utmp0*rorder);
	  utmp0tex += rorder;
	  ttmp0 = tex1D(tex,utmp0tex);
	  mtmp0 += 1;
#ifndef USE_SHARED_FOR_GRADSUM
	  if (mtmp0 >= sizex) mtmp0 -= sizex;
#endif
	  mioffset[0]=mtmp0-miorg[0];//if(mioffset[0]<0) mioffset[0]+=QS_SIZEX;if(mioffset[0]>=QS_SIZEX) mioffset[0]-=QS_SIZEX;
#ifdef USE_SHARED_FOR_GRADSUM
	  qtmp=qs[mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])];
#else
	  qtmp = q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))];
#endif
	  ftmp.x += qtmp*ttmp0.y*ttmp1.x*ttmp2.x;
	  ftmp.y += qtmp*ttmp0.x*ttmp1.y*ttmp2.x;
	  ftmp.z += qtmp*ttmp0.x*ttmp1.x*ttmp2.y;
	  ftmp.w += qtmp*ttmp0.x*ttmp1.x*ttmp2.x;
	}
      }
    }
    ftmp.x *= (float)sizex;
    ftmp.y *= (float)sizey;
    ftmp.z *= (float)sizez;
  }
  f[num].x = xi.w*(ftmp.x*recipf_dev[0]+ftmp.y*recipf_dev[3]+ftmp.z*recipf_dev[6]);
  f[num].y = xi.w*(ftmp.x*recipf_dev[1]+ftmp.y*recipf_dev[4]+ftmp.z*recipf_dev[7]);
  f[num].z = xi.w*(ftmp.x*recipf_dev[2]+ftmp.y*recipf_dev[5]+ftmp.z*recipf_dev[8]);
  f[num].w = xi.w*ftmp.w;
#ifdef MULTIPLE_IBLOCK
  }
#endif
}


extern "C"
__global__ void gpupme1gradsum_kernel_org
(float4* x, float4* f, float* q, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder)
{
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;
  float ui[3], qtmp, utmp1, utmp0;
  float4 ftmp, xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0;
    
  SET_FLOAT4_ZERO(ftmp);
  xi = x[num];
  ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
  ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
  ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#if 1
  for(int j=0;j<3;j++){
    while(ui[j]<0.0f)  ui[j]+=1.0f;
    while(ui[j]>=1.0f) ui[j]-=1.0f;
  }
#endif                             
  ui[0] *= (float)sizex;
  ui[1] *= (float)sizey;
  ui[2] *= (float)sizez;
  mi[0] = (int)ui[0];
  mi[1] = (int)ui[1];
  mi[2] = (int)ui[2];
  ui[0] = - ui[0] + (float)mi[0];
  ui[1] = - ui[1] + (float)mi[1];
  ui[2] = - ui[2] + (float)mi[2];
  mi[0] += sizex - order;
  mi[1] += sizey - order;
  mi[2] += sizez - order;
#if 1 // added to ensure overflow of grids
  if(mi[0]+1>=sizex) mi[0]-=sizex;
  if(mi[1]+1>=sizey) mi[1]-=sizey;
  if(mi[2]+1>=sizez) mi[2]-=sizez;
#endif
#if 1 // subtracted to ensure overflow of grids
  if(mi[0]+1<0) mi[0]+=sizex;
  if(mi[1]+1<0) mi[1]+=sizey;
  if(mi[2]+1<0) mi[2]+=sizez;
#endif           
  float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
  for (int k = 0; k < order; k++){
    ui[2] += 1.f;
    //    ttmp2 = tex1D(tex,ui[2]*rorder);
    ui2tex += rorder;
    ttmp2 = tex1D(tex,ui2tex);
    mi[2] += 1;
    if (mi[2] >= sizez) mi[2] -= sizez;
    utmp1 = ui[1];
    mtmp1 = mi[1];
    float utmp1tex = utmp1*rorder*(NSEG-1)/(float)NSEG;
    for (int j = 0; j < order; j++){
      utmp1 += 1.f;
      //      ttmp1 = tex1D(tex,utmp1*rorder);
      utmp1tex += rorder;
      ttmp1 = tex1D(tex,utmp1tex);
      mtmp1 += 1;
      if (mtmp1 >= sizey) mtmp1 -= sizey;
      utmp0 = ui[0];
      mtmp0 = mi[0];
      float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
      for (int i = 0; i < order; i++){
	utmp0 += 1.f;
	//	ttmp0 = tex1D(tex,utmp0*rorder);
	utmp0tex += rorder;
	ttmp0 = tex1D(tex,utmp0tex);
	mtmp0 += 1;
	if (mtmp0 >= sizex) mtmp0 -= sizex;
	qtmp = q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))];
	ftmp.x += qtmp*ttmp0.y*ttmp1.x*ttmp2.x;
	ftmp.y += qtmp*ttmp0.x*ttmp1.y*ttmp2.x;
	ftmp.z += qtmp*ttmp0.x*ttmp1.x*ttmp2.y;
	ftmp.w += qtmp*ttmp0.x*ttmp1.x*ttmp2.x;
      }
    }
  }
  ftmp.x *= (float)sizex;
  ftmp.y *= (float)sizey;
  ftmp.z *= (float)sizez;
  f[num].x = xi.w*(ftmp.x*recipf_dev[0]+ftmp.y*recipf_dev[3]+ftmp.z*recipf_dev[6]);
  f[num].y = xi.w*(ftmp.x*recipf_dev[1]+ftmp.y*recipf_dev[4]+ftmp.z*recipf_dev[7]);
  f[num].z = xi.w*(ftmp.x*recipf_dev[2]+ftmp.y*recipf_dev[5]+ftmp.z*recipf_dev[8]);
  f[num].w = xi.w*ftmp.w;
}


extern "C"
__global__ void gpupme1gradsum_kernel_simple
(float4* x, float4* f, float* q, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder)
{
  float ui[3], qtmp, utmp1, utmp0;
  float4 ftmp, xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0, mtmp2;
  int i;
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;

  SET_FLOAT4_ZERO(ftmp);
  xi = x[num];
  if(1 && xi.w!=0.0f){
    //  if(1){
#if defined(CELLINDEX) && 1
    ui[0] = __fmul_rn(xi.x,recipf_dev[0])+__fmul_rn(xi.y,recipf_dev[1])+__fmul_rn(xi.z,recipf_dev[2]);
    ui[1] = __fmul_rn(xi.x,recipf_dev[3])+__fmul_rn(xi.y,recipf_dev[4])+__fmul_rn(xi.z,recipf_dev[5]);
    ui[2] = __fmul_rn(xi.x,recipf_dev[6])+__fmul_rn(xi.y,recipf_dev[7])+__fmul_rn(xi.z,recipf_dev[8]);
#else
    ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
    ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
    ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#endif
#if 1
    for(int j=0;j<3;j++){
      while(ui[j]<0.0f)  ui[j]+=1.0f;
      while(ui[j]>=1.0f) ui[j]-=1.0f;
    }
#endif                             
    ui[0] *= (float)sizex;
    ui[1] *= (float)sizey;
    ui[2] *= (float)sizez;
    mi[0] = (int)ui[0];
    mi[1] = (int)ui[1];
    mi[2] = (int)ui[2];
    ui[0] = - ui[0] + (float)mi[0];
    ui[1] = - ui[1] + (float)mi[1];
    ui[2] = - ui[2] + (float)mi[2];
    mi[0] += sizex - order;
    mi[1] += sizey - order;
    mi[2] += sizez - order;
    if(mi[0]+1>=sizex) mi[0]-=sizex;
    if(mi[1]+1>=sizey) mi[1]-=sizey;
    if(mi[2]+1>=sizez) mi[2]-=sizez;
#if 1 // subtracted to ensure overflow of grids
    if(mi[0]+1<0) mi[0]+=sizex;
    if(mi[1]+1<0) mi[1]+=sizey;
    if(mi[2]+1<0) mi[2]+=sizez;
#endif           
    float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
    for (int k = 0; k < order; k++){
      ui[2] += 1.f;
      //      ttmp2 = tex1D(tex,ui[2]*rorder);
      ui2tex += rorder;
      ttmp2 = tex1D(tex,ui2tex);
      mi[2] += 1;
      //      if (mi[2] >= sizez) mi[2] -= sizez;
      mi[2]=mi[2] % sizez;
      utmp1 = ui[1];
      mtmp1 = mi[1];
      float utmp1tex = utmp1*rorder*(NSEG-1)/(float)NSEG;
      for (int j = 0; j < order; j++){
	utmp1 += 1.f;
	//	ttmp1 = tex1D(tex,utmp1*rorder);
	utmp1tex += rorder;
	ttmp1 = tex1D(tex,utmp1tex);
	mtmp1 += 1;
	//	if (mtmp1 >= sizey) mtmp1 -= sizey;
	mtmp1=mtmp1 % sizey;
	utmp0 = ui[0];
	mtmp0 = mi[0];
	float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
	for (int i = 0; i < order; i++){
	  utmp0 += 1.f;
	  //	  ttmp0 = tex1D(tex,utmp0*rorder);
	  utmp0tex += rorder;
	  ttmp0 = tex1D(tex,utmp0tex);
	  mtmp0 += 1;
	  //	  if (mtmp0 >= sizex) mtmp0 -= sizex;
	  mtmp0=mtmp0 % sizex;
	  qtmp = q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))];
	  ftmp.x += qtmp*ttmp0.y*ttmp1.x*ttmp2.x;
	  ftmp.y += qtmp*ttmp0.x*ttmp1.y*ttmp2.x;
	  ftmp.z += qtmp*ttmp0.x*ttmp1.x*ttmp2.y;
	  ftmp.w += qtmp*ttmp0.x*ttmp1.x*ttmp2.x;
	}
      }
    }
    ftmp.x *= (float)sizex;
    ftmp.y *= (float)sizey;
    ftmp.z *= (float)sizez;
  }
  f[num].x = xi.w*(ftmp.x*recipf_dev[0]+ftmp.y*recipf_dev[3]+ftmp.z*recipf_dev[6]);
  f[num].y = xi.w*(ftmp.x*recipf_dev[1]+ftmp.y*recipf_dev[4]+ftmp.z*recipf_dev[7]);
  f[num].z = xi.w*(ftmp.x*recipf_dev[2]+ftmp.y*recipf_dev[5]+ftmp.z*recipf_dev[8]);
  f[num].w = xi.w*ftmp.w;
}


__device__ void gpupme1fillch_kernel_sub
(float4* x, INT_OR_FLOAT *q, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder)
{
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;
  float ui[3], utmp1, utmp0;
  float4 xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0;

  //  if(num>=BSIZEPME2) return;
  //  if(num>0) return;
  //  if(!(num>=BSIZEPME2*7 && num<BSIZEPME2*8)) return;
  //  if(!(num>=BSIZEPME2*0 && num<BSIZEPME2*1)) return;
  xi = x[num];
  //  if(threadIdx.x!=14) return;
  ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
  ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
  ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#if 1
  for(int j=0;j<3;j++){
    while(ui[j]<0.0f)  ui[j]+=1.0f;
    while(ui[j]>=1.0f) ui[j]-=1.0f;
  }
#endif                             
  ui[0] *= (float)sizex;
  ui[1] *= (float)sizey;
  ui[2] *= (float)sizez;
  mi[0] = (int)ui[0];
  mi[1] = (int)ui[1];
  mi[2] = (int)ui[2];
  ui[0] = - ui[0] + (float)mi[0];
  ui[1] = - ui[1] + (float)mi[1];
  ui[2] = - ui[2] + (float)mi[2];
  mi[0] += sizex - order;
  mi[1] += sizey - order;
  mi[2] += sizez - order;
#if 1 // added to ensure overflow of grids
  if(mi[0]+1>=sizex) mi[0]-=sizex;
  if(mi[1]+1>=sizey) mi[1]-=sizey;
  if(mi[2]+1>=sizez) mi[2]-=sizez;
#endif
#if 1 // subtracted to ensure overflow of grids
  if(mi[0]+1<0) mi[0]+=sizex;
  if(mi[1]+1<0) mi[1]+=sizey;
  if(mi[2]+1<0) mi[2]+=sizez;
#endif           
  float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
  for (int k = 0; k < order; k++){
    //  for (int k = 0; k < 1; k++){
    ui[2] += 1.f;
    //    ttmp2 = tex1D(tex,ui[2]*rorder);
    ui2tex += rorder;
    ttmp2 = tex1D(tex,ui2tex);
    mi[2] += 1;
    if (mi[2] >= sizez) mi[2] -= sizez;
    utmp1 = ui[1];
    mtmp1 = mi[1];
    float utmp1tex = utmp1*rorder*(NSEG-1)/(float)NSEG;
    for (int j = 0; j < order; j++){
      //    for (int j = 0; j < 2; j++){
      utmp1 += 1.f;
      //      ttmp1 = tex1D(tex,utmp1*rorder);
      utmp1tex += rorder;
      ttmp1 = tex1D(tex,utmp1tex);
      mtmp1 += 1;
      if (mtmp1 >= sizey) mtmp1 -= sizey;
      utmp0 = ui[0];
      mtmp0 = mi[0];
      float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
      for (int i = 0; i < order; i++){
	//      for (int i = 0; i < 2; i++){
	utmp0 += 1.f;
	//	ttmp0 = tex1D(tex,utmp0*rorder);
	utmp0tex += rorder;
	ttmp0 = tex1D(tex,utmp0tex);
	mtmp0 += 1;
	if (mtmp0 >= sizex) mtmp0 -= sizex;
	//	q[(0+i+j*2)*2]=mtmp0;q[(4+i+j*2)*2]=mtmp1;q[(8+i+j*2)*2]=mi[2];q[12*2]=sizex;q[13*2]=sizey;q[14*2]=sizez;
	//        printf("threadIdx.x=%d mtmp0=%d mtmp1=%d mi[2]=%d\n",threadIdx.x,mtmp0,mtmp1,mi[2]);
	atomicAdd(&q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))],
#ifdef FIXEDPOINT
		  (int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT)
#else
		  xi.w*ttmp0.x*ttmp1.x*ttmp2.x
#endif
		  );
      }
    }
  }
}

extern "C"
__global__ void gpupme1fillch_kernel
(float4* x, INT_OR_FLOAT *q, INT_OR_FLOAT *q2, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder,
 int (*cpb)[4], int gridperblockx, int gridperblocky, int gridperblockz)
{
  float ui[3], utmp1, utmp0;
  float4 xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0, mtmp2;
  __shared__ INT_OR_FLOAT qs[QS_SIZE];
  //  if(blockIdx.x==0 && threadIdx.x==0) printf("gridperblock=%d %d %d\n",gridperblockx,gridperblocky,gridperblockz);
#if defined(MULTIPLE_IBLOCK_TEST2) && 0
  //  int ncc=cpb[blockIdx.x][1]>>8;
  int bidx2=cpb[blockIdx.x][0]>>8;
#if 0
  if(ncc<=0){
    if(threadIdx.x==0) printf("bid=%d ncc=%d bidx2=%d\n",blockIdx.x,ncc,bidx2);
    return;
  }
#endif
  int cpt=cpb[bidx2][3] & 0xf;
  int cpbone[3]={(cpb[bidx2][0] & 0xff)*gridperblockx,
		 (cpb[bidx2][1] & 0xff)*gridperblocky,
		 (cpb[bidx2][2] & 0xff)*gridperblockz};
#else
  int cpt=cpb[blockIdx.x][3] & 0xf;
  int cpbone[3]={(cpb[blockIdx.x][0] & 0xff)*gridperblockx,
		 (cpb[blockIdx.x][1] & 0xff)*gridperblocky,
		 (cpb[blockIdx.x][2] & 0xff)*gridperblockz};
#endif
  int i,cptoffset=cpt*sizex*sizey*sizez*2;
  int miorg[3],mioffset[3];
#ifdef MULTIPLE_IBLOCK
  int fb=cpb[blockIdx.x][0]>>8,nb=(cpb[blockIdx.x][1]>>8)-fb;
  int num = fb * BSIZEPME2 + threadIdx.x;
#elif defined(MULTIPLE_IBLOCK_TEST2) && 0
  int num = bidx2 * BSIZEPME2 + threadIdx.x;
#else
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;
#endif

#if 0
  if(num>=4*BSIZEPME2) return;
#endif
#if 0
  if(blockIdx.x!=404) return;
#endif
#ifndef MULTIPLE_IBLOCK_TEST2
  if(cpt>=8){ // normal calculation with global atomicadd
#ifndef MULTIPLE_IBLOCK_TEST
    gpupme1fillch_kernel_sub(x,q,sizex,sizey,sizez,
                             recipf_dev,order,rorder);
#endif
    //    if(threadIdx.x==0) printf("cpt=%d>=8 bid=%d\n",cpt,blockIdx.x);
    return;
  }
#endif // end of not MULTIPLE_IBLOCK_TEST2

  for(i=threadIdx.x;i<QS_SIZE;i+=BSIZEPME2) qs[i]=0;
  __syncthreads();

  for(int k=0;k<3;k++) miorg[k]=cpbone[k];
#ifdef MULTIPLE_IBLOCK
  for(int b=0;b<nb;b++,num+=BSIZEPME2){
#elif defined(MULTIPLE_IBLOCK_TEST)
#if defined(MULTIPLE_IBLOCK_TEST2) && 0
  for(int b=0;b<(cpb[bidx2][3]>>4);b++,num+=BSIZEPME2){
#else
  for(int b=0;b<(cpb[blockIdx.x][3]>>4);b++,num+=BSIZEPME2){
#endif
#endif
  xi = x[num];
  //  if(threadIdx.x!=14) goto skip_loop;
  if(1 && xi.w!=0.0f){
#if defined(CELLINDEX) && 1
    ui[0] = __fmul_rn(xi.x,recipf_dev[0])+__fmul_rn(xi.y,recipf_dev[1])+__fmul_rn(xi.z,recipf_dev[2]);
    ui[1] = __fmul_rn(xi.x,recipf_dev[3])+__fmul_rn(xi.y,recipf_dev[4])+__fmul_rn(xi.z,recipf_dev[5]);
    ui[2] = __fmul_rn(xi.x,recipf_dev[6])+__fmul_rn(xi.y,recipf_dev[7])+__fmul_rn(xi.z,recipf_dev[8]);
#else
    ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
    ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
    ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#endif
#if 1
    for(int j=0;j<3;j++){
      while(ui[j]<0.0f)  ui[j]+=1.0f;
      while(ui[j]>=1.0f) ui[j]-=1.0f;
    }
#endif                             
    ui[0] *= (float)sizex;
    ui[1] *= (float)sizey;
    ui[2] *= (float)sizez;
    mi[0] = (int)ui[0];
    mi[1] = (int)ui[1];
    mi[2] = (int)ui[2];
    ui[0] = - ui[0] + (float)mi[0];
    ui[1] = - ui[1] + (float)mi[1];
    ui[2] = - ui[2] + (float)mi[2];
    mi[0] += sizex - order;
    mi[1] += sizey - order;
    mi[2] += sizez - order;
#if defined(CELLINDEX) && 1
    int qoffset;
    qoffset=miorg[0]+sizex*(miorg[1]+sizey*miorg[2]);
#endif

#if 1 // minimum image conversion
    while(mi[0]-miorg[0]>=sizex/2) mi[0]-=sizex;
    while(mi[1]-miorg[1]>=sizey/2) mi[1]-=sizey;
    while(mi[2]-miorg[2]>=sizez/2) mi[2]-=sizez;
    while(mi[0]-miorg[0]<-sizex/2) mi[0]+=sizex;
    while(mi[1]-miorg[1]<-sizey/2) mi[1]+=sizey;
    while(mi[2]-miorg[2]<-sizez/2) mi[2]+=sizez;
#else // not complete    
    if(mi[0]+1>=sizex) mi[0]-=sizex;
    if(mi[1]+1>=sizey) mi[1]-=sizey;
    if(mi[2]+1>=sizez) mi[2]-=sizez;
    if(mi[0]+1<0)        mi[0]+=sizex;
    if(mi[1]+1<0)        mi[1]+=sizey;
    if(mi[2]+1<0)        mi[2]+=sizez;
#endif
    if(num==BSIZEPME2*8-1 && 0){
      q2[6]=miorg[0];
      q2[8]=miorg[1];
      q2[10]=miorg[2];
      q2[12]=mi[0];
      q2[14]=mi[1];
      q2[16]=mi[2];
    }
    
    float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
    for (int k = 0; k < order; k++){
      //    for (int k = 0; k < 1; k++){
    //for (int k = 0; k < order && num==BSIZEPME2*8-1; k++){
    //  for (int k = 0; k < order && num>=BSIZEPME2*7 && num<BSIZEPME2*8; k++){
    //  for (int k = 0; k < order && num>=BSIZEPME2*0 && num<BSIZEPME2*1; k++){
    //    for (int k = 0; k < 1 && num==0; k++){
      ui[2] += 1.f;
      //      ttmp2 = tex1D(tex,ui[2]*rorder);
      ui2tex += rorder;
      ttmp2 = tex1D(tex,ui2tex);
      mi[2] += 1;
      //    if (mi[2] >= sizez) mi[2] -= sizez;
      mioffset[2]=mi[2]-miorg[2];//if(mioffset[2]<0) mioffset[2]+=sizez;if(mioffset[2]>=sizez) mioffset[2]-=sizez;
      utmp1 = ui[1];
      mtmp1 = mi[1];
      float utmp1tex = utmp1*rorder*(NSEG-1)/(float)NSEG;
      for (int j = 0; j < order; j++){
	//for (int j = 0; j < 2; j++){
	utmp1 += 1.f;
	//	ttmp1 = tex1D(tex,utmp1*rorder);
	utmp1tex += rorder;
	ttmp1 = tex1D(tex,utmp1tex);
	mtmp1 += 1;
	//	if (mtmp1 >= sizey) mtmp1 -= sizey;
	mioffset[1]=mtmp1-miorg[1];//if(mioffset[1]<0) mioffset[1]+=sizey;if(mioffset[1]>=sizey) mioffset[1]-=sizey;
	utmp0 = ui[0];
	mtmp0 = mi[0];
	float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
	for (int i = 0; i < order; i++){
	  //	for (int i = 0; i < 2; i++){
	  utmp0 += 1.f;
	  //	  ttmp0 = tex1D(tex,utmp0*rorder);
	  utmp0tex += rorder;
	  ttmp0 = tex1D(tex,utmp0tex);
	  mtmp0 += 1;
	  //	if (mtmp0 >= sizex) mtmp0 -= sizex;
	  mioffset[0]=mtmp0-miorg[0];//if(mioffset[0]<0) mioffset[0]+=sizex;if(mioffset[0]>=sizex) mioffset[0]-=sizex;
	  //	mtmp0 = mtmp0 % sizex;
	  
	  //	  atomicAdd(&q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))],(int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT));
	  //	atomicAdd(&q2[cptoffset+2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))],(int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT));
	  //	printf("cptoffset=%d %d\n",cptoffset,cptoffset/(sizex*sizey*sizez*2));
	  
	  //	atomicAdd(&q2[cptoffset+2*(qoffset+mioffset[0]+sizex*(mioffset[1]+sizey*mioffset[2]))],(int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT));
	  //	  q[(0+i+j*2)*2]=mtmp0;q[(4+i+j*2)*2]=mtmp1;q[(8+i+j*2)*2]=mi[2];q[12*2]=miorg[0];q[13*2]=miorg[1];q[14*2]=miorg[2];
	  //	  q[(0+i+j*2)*2]=mtmp0;q[(4+i+j*2)*2]=mtmp1;q[(8+i+j*2)*2]=mi[2];
	  //	  q[(0+i+j*2)*2]=mioffset[0];q[(4+i+j*2)*2]=mioffset[1];q[(8+i+j*2)*2]=mioffset[2];q[12*2]=miorg[0];q[13*2]=miorg[1];q[14*2]=miorg[2];q[15*2]=mi[0];q[16*2]=mi[1];q[17*2]=mi[2];
	  //	  printf("threadIdx.x=%d mtmp0=%d mtmp1=%d mi[2]=%d\n",threadIdx.x,mtmp0,mtmp1,mi[2]);
	  //	  printf("threadIdx.x=%d mioffset[0]=%d mioffset[1]=%d mioffset[2]=%d miorg=%d %d %d\n",threadIdx.x,mioffset[0],mioffset[1],mioffset[2],miorg[0],miorg[1],miorg[2]);

#if 0
	  //	  if(mioffset[0]<0) mioffset[0]+=QS_SIZEX;if(mioffset[0]>=QS_SIZEX) mioffset[0]-=QS_SIZEX;
	  //	  if(mioffset[1]<0) mioffset[1]+=QS_SIZEY;if(mioffset[1]>=QS_SIZEY) mioffset[1]-=QS_SIZEY;
	  //	  if(mioffset[2]<0) mioffset[2]+=QS_SIZEZ;if(mioffset[2]>=QS_SIZEZ) mioffset[2]-=QS_SIZEZ;
	  if(i==0 && j==0 && k==0 && blockIdx.x==0 && threadIdx.x==0) printf("** debug message in fillch_kernel **\n");
	  if(mtmp0+sizex*(mtmp1+sizey*mi[2])==47 ||
	     //	     xi.x==47.51550f || 
	     //	     xi.x==-12.74828f ||
	     //	     xi.x==46.56997f || 
	     //	     xi.x==-14.80412f ||
	     xi.x==-13.15800f || 
	     //	     xi.x==-13.60596f ||
	     //	     xi.x==-14.96458f || 
	     //	     xi.x==-14.28743f ||
	     0){
	    float uitmp[3];
	    int mitmp[3];
	    uitmp[0] = __fmul_rn(xi.x,recipf_dev[0])+__fmul_rn(xi.y,recipf_dev[1])+__fmul_rn(xi.z,recipf_dev[2]);
	    uitmp[1] = __fmul_rn(xi.x,recipf_dev[3])+__fmul_rn(xi.y,recipf_dev[4])+__fmul_rn(xi.z,recipf_dev[5]);
	    uitmp[2] = __fmul_rn(xi.x,recipf_dev[6])+__fmul_rn(xi.y,recipf_dev[7])+__fmul_rn(xi.z,recipf_dev[8]);
	    for(int j=0;j<3;j++){
	      while(uitmp[j]<0.0f)  uitmp[j]+=1.0f;
	      while(uitmp[j]>=1.0f) uitmp[j]-=1.0f;
	    }
	    uitmp[0] *= (float)sizex;
	    uitmp[1] *= (float)sizey;
	    uitmp[2] *= (float)sizez;
	    mitmp[0] = (int)uitmp[0];
	    mitmp[1] = (int)uitmp[1];
	    mitmp[2] = (int)uitmp[2];
	    uitmp[0] = - uitmp[0] + (float)mitmp[0];
	    uitmp[1] = - uitmp[1] + (float)mitmp[1];
	    uitmp[2] = - uitmp[2] + (float)mitmp[2];
	    mitmp[0] += sizex - order;
	    mitmp[1] += sizey - order;
	    mitmp[2] += sizez - order;
#if 1 // minimum image conversion
	    while(mitmp[0]-miorg[0]>=sizex/2) mitmp[0]-=sizex;
	    while(mitmp[1]-miorg[1]>=sizey/2) mitmp[1]-=sizey;
	    while(mitmp[2]-miorg[2]>=sizez/2) mitmp[2]-=sizez;
	    while(mitmp[0]-miorg[0]<-sizex/2) mitmp[0]+=sizex;
	    while(mitmp[1]-miorg[1]<-sizey/2) mitmp[1]+=sizey;
	    while(mitmp[2]-miorg[2]<-sizez/2) mitmp[2]+=sizez;
#endif
	    printf(" bid=%d tid=%d num=%d mtmp0=%d mtmp1=%d mi[2]=%d miorg=%d %d %d mioffset=%d %d %d\n  xi[%d]=%e %e %e uitmp=%e %e %e mitmp=%d %d %d\n                   ui=%e %e %e mi=%d %d %d   qsdiff=%d %e\n",
		   blockIdx.x,threadIdx.x,num,mtmp0,mtmp1,mi[2],miorg[0],miorg[1],miorg[2],mioffset[0],mioffset[1],mioffset[2],
		   num,xi.x,xi.y,xi.z,uitmp[0],uitmp[1],uitmp[2],mitmp[0],mitmp[1],mitmp[2],
		   ui[0],ui[1],ui[2],mi[0],mi[1],mi[2],(int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT),(int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT)/FIXEDPOINT);
	  }
#if 0
	  if(mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])<0 || mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])>=QS_SIZE){
	    printf("bid=%d tid=%d mioffset=%d %d %d qs[%d]\n",blockIdx.x,threadIdx.x,mioffset[0],mioffset[1],mioffset[2],mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2]));
	  }
#endif
	  if(mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])>=0 && mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])<QS_SIZE)
#endif
	  atomicAdd(&qs[mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])],
#ifdef FIXEDPOINT
		    (int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT)   // work
#else
		    xi.w*ttmp0.x*ttmp1.x*ttmp2.x
#endif
		    );
	  if(0 && 
	     (mioffset[0]+QS_SIZEX*(mioffset[1]+QS_SIZEY*mioffset[2])>=QS_SIZE 
	      || mioffset[0]>=QS_SIZEX || mioffset[1]>=QS_SIZEY 
	      || mioffset[2]>=QS_SIZEZ
	      || mioffset[0]<0 || mioffset[1]<0 || mioffset[2]<0)){
	    q[0]=-999999;
	    q[2]=mioffset[0];
	    q[4]=mioffset[1];
	    q[6]=mioffset[2];
	    q[8]=num;
	  }
	  if(0 && num>=BSIZEPME2*7 && num<BSIZEPME2*8){
	    q[2*order*order*order*3+2*(3*(i+order*(j+order*k)))]=mioffset[0];
	    q[2*order*order*order*3+2*(3*(i+order*(j+order*k))+1)]=mioffset[1];
	    q[2*order*order*order*3+2*(3*(i+order*(j+order*k))+2)]=mioffset[2];
	  }
	}
      }
    }
  }
#if defined(MULTIPLE_IBLOCK) || defined(MULTIPLE_IBLOCK_TEST)
  }
#endif

 skip_loop:;    
  __syncthreads();
#if 0
  for(i=threadIdx.x;i<sizex*sizey*sizez;i+=BSIZEPME2){
    mtmp0=i % sizex;
    mtmp1=(i/sizex) % sizey;
    mtmp2=(i/sizex/sizey) % sizez;
    mtmp0+=miorg[0];if(mtmp0>=sizex) mtmp0-=sizex;
    mtmp1+=miorg[1];if(mtmp1>=sizey) mtmp1-=sizey;
    mtmp2+=miorg[2];if(mtmp2>=sizez) mtmp2-=sizez;
    q2[cptoffset+2*(mtmp0+sizex*(mtmp1+sizey*mtmp2))]=qs[i];
  }
#else // work
  for(i=threadIdx.x;i<QS_SIZE;i+=BSIZEPME2){
    mtmp0=i % QS_SIZEX;
    mtmp1=(i/QS_SIZEX) % QS_SIZEY;
    mtmp2=(i/QS_SIZEX/QS_SIZEY) % QS_SIZEZ;
    mtmp0+=miorg[0];if(mtmp0>=sizex) mtmp0-=sizex;
    mtmp1+=miorg[1];if(mtmp1>=sizey) mtmp1-=sizey;
    mtmp2+=miorg[2];if(mtmp2>=sizez) mtmp2-=sizez;
    q2[cptoffset+2*(mtmp0+sizex*(mtmp1+sizey*mtmp2))]=qs[i];
#if 0
    if(cptoffset+2*(mtmp0+sizex*(mtmp1+sizey*mtmp2))==2){
      printf("bid=%d tid=%d i=%d mtmp0=%d mtmp1=%d mtmp2=%d cptoffset=%d qs=%d\n",
	     blockIdx.x,threadIdx.x,i,mtmp0,mtmp1,mtmp2,cptoffset,qs[i]);
    }
#endif
  }
#endif
  __syncthreads();
}

extern "C"
__global__ void gpupme1fillch_kernel_org
(float4* x, INT_OR_FLOAT *q, int sizex, int sizey, int sizez,
 float* recipf_dev, int order, float rorder)
{
  int num = blockIdx.x * BSIZEPME2 + threadIdx.x;
  float ui[3], utmp1, utmp0;
  float4 xi;
  float2 ttmp0, ttmp1, ttmp2;
  int mi[3], mtmp1, mtmp0;
    
  xi = x[num];
  ui[0] = xi.x*recipf_dev[0]+xi.y*recipf_dev[1]+xi.z*recipf_dev[2];
  ui[1] = xi.x*recipf_dev[3]+xi.y*recipf_dev[4]+xi.z*recipf_dev[5];
  ui[2] = xi.x*recipf_dev[6]+xi.y*recipf_dev[7]+xi.z*recipf_dev[8];
#if 1
  for(int j=0;j<3;j++){
    while(ui[j]<0.0f)  ui[j]+=1.0f;
    while(ui[j]>=1.0f) ui[j]-=1.0f;
  }
#endif                             
  ui[0] *= (float)sizex;
  ui[1] *= (float)sizey;
  ui[2] *= (float)sizez;
  mi[0] = (int)ui[0];
  mi[1] = (int)ui[1];
  mi[2] = (int)ui[2];
  ui[0] = - ui[0] + (float)mi[0];
  ui[1] = - ui[1] + (float)mi[1];
  ui[2] = - ui[2] + (float)mi[2];
  mi[0] += sizex - order;
  mi[1] += sizey - order;
  mi[2] += sizez - order;
#if 1 // added to ensure overflow of grids
  if(mi[0]+1>=sizex) mi[0]-=sizex;
  if(mi[1]+1>=sizey) mi[1]-=sizey;
  if(mi[2]+1>=sizez) mi[2]-=sizez;
#endif
#if 1 // subtracted to ensure overflow of grids
  if(mi[0]+1<0) mi[0]+=sizex;
  if(mi[1]+1<0) mi[1]+=sizey;
  if(mi[2]+1<0) mi[2]+=sizez;
#endif           
#if 0
  if(num==2325 || num==6097 || num==2322 || num==2342 || 
     num==1799 || num==6095 || num==2341 || num==2323){
    printf("  xi[%d]=%e %e %e ui=%e %e %e mi=%d %d %d\n",
	   num,xi.x,xi.y,xi.z,ui[0],ui[1],ui[2],
	   mi[0],mi[1],mi[2]);
  }
#endif
  float ui2tex = ui[2]*rorder*(NSEG-1)/(float)NSEG;
  for (int k = 0; k < order; k++){
    //  for (int k = 0; k < 1; k++){
    ui[2] += 1.f;
    //    ttmp2 = tex1D(tex,ui[2]*rorder);
    ui2tex += rorder;
    ttmp2 = tex1D(tex,ui2tex);
    mi[2] += 1;
    if (mi[2] >= sizez) mi[2] -= sizez;
    utmp1 = ui[1];
    mtmp1 = mi[1];
    float utmp1tex = utmp1*rorder*(NSEG-1)/(float)NSEG;
    for (int j = 0; j < order; j++){
      //    for (int j = 0; j < 2; j++){
      utmp1 += 1.f;
      //      ttmp1 = tex1D(tex,utmp1*rorder);
      utmp1tex += rorder;
      ttmp1 = tex1D(tex,utmp1tex);
      mtmp1 += 1;
      if (mtmp1 >= sizey) mtmp1 -= sizey;
      utmp0 = ui[0];
      mtmp0 = mi[0];
      float utmp0tex = utmp0*rorder*(NSEG-1)/(float)NSEG;
      for (int i = 0; i < order; i++){
	//      for (int i = 0; i < 2; i++){
	utmp0 += 1.f;
	//	ttmp0 = tex1D(tex,utmp0*rorder);
	utmp0tex += rorder;
	ttmp0 = tex1D(tex,utmp0tex);
	mtmp0 += 1;
	if (mtmp0 >= sizex) mtmp0 -= sizex;
#if 0
	if(i==0 && j==0 && k==0 && blockIdx.x==0 && threadIdx.x==0) printf("** debug in fillch_kernel **\n");
	if(mtmp0+sizex*(mtmp1+sizey*mi[2])==47){
	  printf(" bid=%d tid=%d num=%d mtmp0=%d mtmp1=%d mi[2]=%d q[%d]=%d + %d(%e)\n",
		 blockIdx.x,threadIdx.x,
		 num,mtmp0,mtmp1,mi[2],
		 mtmp0+sizex*(mtmp1+sizey*mi[2]),
		 q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))],
		 (int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT),
		 (int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT)/FIXEDPOINT);
	}
#endif
	atomicAdd(&q[2*(mtmp0+sizex*(mtmp1+sizey*mi[2]))],
#ifdef FIXEDPOINT
		  (int)(xi.w*ttmp0.x*ttmp1.x*ttmp2.x*FIXEDPOINT)
#else
		  xi.w*ttmp0.x*ttmp1.x*ttmp2.x
#endif
		  );
      }
    }
  }
}

extern "C"
__global__ void gpupme1exchange_kernel
(INT_OR_FLOAT *qm, INT_OR_FLOAT *qm2, float* q_dev, int sizex, int sizey, int sizez)
{
  //int nx = blockIdx.x * BSIZEPME1 + threadIdx.x;
  //int ny = blockIdx.y * BSIZEPME1 + threadIdx.y;
  //int nz = blockIdx.z * BSIZEPME1 + threadIdx.z;
  //int num = nx + ny * sizex + nz * sizex * sizey;
#if 1 // modified by Ryuji, Nov. 2011
  int num = (blockIdx.y+blockIdx.x*gridDim.y) * BSIZEPME1 + threadIdx.x;
  if (num >= sizex*sizey*sizez) return;
#else
  int num = blockIdx.x * BSIZEPME1 + threadIdx.x;
#endif
#if defined(CELLINDEX) && 1
#if 1 // sum up by integer
  INT_OR_FLOAT sum=qm[num*2+0];
  for(int i=0;i<8;i++){
#if 0
    int memsize_qm2=sizeof(INT_OR_FLOAT)*sizex*sizey*sizez*2*8;
    if((&qm2[i*sizex*sizey*sizez*2+num*2+0])-(&qm2[0])>=memsize_qm2/sizeof(INT_OR_FLOAT)){
      printf("bid=%d tid=%d num=%d qm2[%d]\n",blockIdx.x,threadIdx.x,num,i*sizex*sizey*sizez*2+num*2+0);
    }
#endif
    sum += qm2[i*sizex*sizey*sizez*2+num*2+0];
  }
#ifdef FIXEDPOINT
  q_dev[num*2+0] = sum/FIXEDPOINT;
#else
  q_dev[num*2+0] = sum;
#endif
  q_dev[num*2+1] = 0.f;
#else // sum up by float
#ifdef FIXEDPOINT
  q_dev[num*2+0] = qm2[num*2+0]/FIXEDPOINT;
#else
  q_dev[num*2+0] = qm2[num*2+0];
#endif
  q_dev[num*2+1] = 0.f;
  for(int i=1;i<8;i++){
#ifdef FIXEDPOINT
    q_dev[num*2+0] += qm2[i*sizex*sizey*sizez*2+num*2+0]/FIXEDPOINT;
#else
    q_dev[num*2+0] += qm2[i*sizex*sizey*sizez*2+num*2+0];
#endif
  }
#endif // end of sum up by int or float
#else // else of CELLINDEX
#ifdef FIXEDPOINT
  q_dev[num*2+0] = qm[num*2+0]/FIXEDPOINT;
#else
  q_dev[num*2+0] = qm[num*2+0];
#endif
  q_dev[num*2+1] = 0.f;
#endif // end of CELLINDEX
}

extern "C"
__global__ void gpupme1initializeq_kernel
(INT_OR_FLOAT *q_dev, INT_OR_FLOAT *q2_dev, int sizex, int sizey, int sizez)
{
#if 1 // modified by Ryuji, Nov. 2011
  int num = (blockIdx.y + blockIdx.x * gridDim.y) * BSIZEPME1 + threadIdx.x;
  if (num >= sizex*sizey*sizez) return;
#else
  int num = blockIdx.x * BSIZEPME1 + threadIdx.x;
#endif
  int offset=sizex*sizey*sizez*2;
  q_dev[num*2+0] = 0;
  q_dev[num*2+1] = 0;
  for(int i=0;i<8;i++){
    __syncthreads();
    q2_dev[i*offset+num*2+0]=q2_dev[i*offset+num*2+1]=0;
  }
}

extern "C"
__global__ void gpupme1exchange_kernel_org
(INT_OR_FLOAT *qm, float* q_dev, int sizex, int sizey, int sizez)
{
  //int nx = blockIdx.x * BSIZEPME1 + threadIdx.x;
  //int ny = blockIdx.y * BSIZEPME1 + threadIdx.y;
  //int nz = blockIdx.z * BSIZEPME1 + threadIdx.z;
  //int num = nx + ny * sizex + nz * sizex * sizey;
#if 1 // modified by Ryuji, Nov. 2011
  int num = (blockIdx.y + blockIdx.x * gridDim.y) * BSIZEPME1 + threadIdx.x;
  if (num >= sizex*sizey*sizez) return;
#else
  int num = blockIdx.x * BSIZEPME1 + threadIdx.x;
#endif
#ifdef FIXEDPOINT
  q_dev[num*2+0] = qm[num*2+0]/FIXEDPOINT;
#else
  q_dev[num*2+0] = qm[num*2+0];
#endif
  q_dev[num*2+1] = 0.f;
}
